#include "hip/hip_runtime.h"
#include <stdio.h>

#include "radonusfft.cuh"
#include "kernels_tomo.cu"
#include "shift.cu"
#include "filter.cu"

radonusfft::radonusfft(size_t ntheta, size_t pnz, size_t n, float center,
                       size_t theta_, size_t ngpus)
    : ntheta(ntheta), pnz(pnz), n(n), center(center), ngpus(ngpus) {
  float eps = 1e-2;
  mu = -log(eps) / (2 * n * n);
  m = ceil(2 * n * 1 / PI * sqrt(-mu * log(eps) + (mu * n) * (mu * n) / 4));
  f = new float2*[ngpus];
  g = new float2*[ngpus];
  fdee = new float2*[ngpus];
  x = new float*[ngpus];
  y = new float*[ngpus];
  shiftfwd = new float2*[ngpus];
  shiftadj = new float2*[ngpus];
  theta = new float*[ngpus];
  plan1d = new hipfftHandle[ngpus];  
  plan2d = new hipfftHandle[ngpus];
  
  for (int igpu=0;igpu<ngpus;igpu++)
  {
    hipSetDevice(igpu);
    hipMalloc((void **)&f[igpu], n * n * pnz * sizeof(float2));
    hipMalloc((void **)&g[igpu], n * ntheta * pnz * sizeof(float2));
    hipMalloc((void **)&fdee[igpu],
              (2 * n + 2 * m) * (2 * n + 2 * m) * pnz * sizeof(float2));

    hipMalloc((void **)&x[igpu], n * ntheta * sizeof(float));
    hipMalloc((void **)&y[igpu], n * ntheta * sizeof(float));
    hipMalloc((void **)&theta[igpu], ntheta * sizeof(float));
    hipMemcpy(theta[igpu], (float *)theta_, ntheta * sizeof(float), hipMemcpyDefault);
    
    int ffts[2];
    int idist;
    int inembed[2];
    // fft 2d
    ffts[0] = 2 * n;
    ffts[1] = 2 * n;
    idist = (2 * n + 2 * m) * (2 * n + 2 * m);
    inembed[0] = 2 * n + 2 * m;
    inembed[1] = 2 * n + 2 * m;
    hipfftPlanMany(&plan2d[igpu], 2, ffts, inembed, 1, idist, inembed, 1, idist,
                  HIPFFT_C2C, pnz);
    
    // fft 1d
    ffts[0] = n;
    idist = n;
    inembed[0] = n;
    hipfftPlanMany(&plan1d[igpu], 1, ffts, inembed, 1, idist, inembed, 1, idist,
                  HIPFFT_C2C, ntheta * pnz);
    hipMalloc((void **)&shiftfwd[igpu], n * sizeof(float2));
    hipMalloc((void **)&shiftadj[igpu], n * sizeof(float2));
    // compute shifts with respect to the rotation center
    takeshift <<<ceil(n / 1024.0), 1024>>> (shiftfwd[igpu], -(center - n / 2.0), n);
    takeshift <<<ceil(n / 1024.0), 1024>>> (shiftadj[igpu], (center - n / 2.0), n);


  }

  //back tp 0
  hipSetDevice(0);


  BS2d = dim3(32, 32);
  BS3d = dim3(32, 32, 1);

  GS2d0 = dim3(ceil(n / (float)BS2d.x), ceil(ntheta / (float)BS2d.y));
  GS3d0 = dim3(ceil(n / (float)BS3d.x), ceil(n / (float)BS3d.y),
              ceil(pnz / (float)BS3d.z));
  GS3d1 = dim3(ceil(2 * n / (float)BS3d.x), ceil(2 * n / (float)BS3d.y),
              ceil(pnz / (float)BS3d.z));
  GS3d2 = dim3(ceil((2 * n + 2 * m) / (float)BS3d.x),
              ceil((2 * n + 2 * m) / (float)BS3d.y), ceil(pnz / (float)BS3d.z));
  GS3d3 = dim3(ceil(n / (float)BS3d.x), ceil(ntheta / (float)BS3d.y),
              ceil(pnz / (float)BS3d.z));
  
}

// destructor, memory deallocation
radonusfft::~radonusfft() { free(); }

void radonusfft::free() {
  if (!is_free) {
    for(int igpu=0;igpu<ngpus;igpu++)
    {
      hipSetDevice(igpu);
      hipFree(f[igpu]);
      hipFree(g[igpu]);
      hipFree(fdee[igpu]);
      hipFree(x[igpu]);
      hipFree(y[igpu]);
      hipFree(shiftfwd[igpu]);
      hipFree(shiftadj[igpu]);
      hipfftDestroy(plan2d[igpu]);
      hipfftDestroy(plan1d[igpu]);
    }
    hipFree(f);
    hipFree(g);
    hipFree(fdee);
    hipFree(x);
    hipFree(y);
    hipFree(shiftfwd);
    hipFree(shiftadj);   
    is_free = true;   
  }
}

void radonusfft::fwd(size_t g_, size_t f_, size_t igpu) {

    hipSetDevice(igpu);
    float2* f0 = (float2 *)f_;
    hipMemcpy(f[igpu], f0, n * n * pnz * sizeof(float2), hipMemcpyDefault);      
    hipMemset(fdee[igpu], 0, (2 * n + 2 * m) * (2 * n + 2 * m) * pnz * sizeof(float2));

    //circ <<<GS3d0, BS3d>>> (f, 1.0f / n, n, pnz);
    takexy <<<GS2d0, BS2d>>> (x[igpu], y[igpu], theta[igpu], n, ntheta);

    divphi <<<GS3d2, BS3d>>> (fdee[igpu], f[igpu], mu, n, pnz, m, TOMO_FWD);
    fftshiftc <<<GS3d2, BS3d>>> (fdee[igpu], 2 * n + 2 * m, pnz);
    hipfftExecC2C(plan2d[igpu], (hipfftComplex *)&fdee[igpu][m + m * (2 * n + 2 * m)],
                (hipfftComplex *)&fdee[igpu][m + m * (2 * n + 2 * m)], HIPFFT_FORWARD);
    fftshiftc <<<GS3d2, BS3d>>> (fdee[igpu], 2 * n + 2 * m, pnz);

    wrap <<<GS3d2, BS3d>>> (fdee[igpu], n, pnz, m, TOMO_FWD);
    gather <<<GS3d3, BS3d>>> (g[igpu], fdee[igpu], x[igpu], y[igpu], m, mu, n, ntheta, pnz, TOMO_FWD);
    // shift with respect to given center
    shift <<<GS3d3, BS3d>>> (g[igpu], shiftfwd[igpu], n, ntheta, pnz);

    ifftshiftc <<<GS3d3, BS3d>>> (g[igpu], n, ntheta, pnz);
    hipfftExecC2C(plan1d[igpu], (hipfftComplex *)g[igpu], (hipfftComplex *)g[igpu], HIPFFT_BACKWARD);
    ifftshiftc <<<GS3d3, BS3d>>> (g[igpu], n, ntheta, pnz);
    if(n%4!=0)
      ifftshiftcmul <<<GS3d3, BS3d>>> (g[igpu], n, ntheta, pnz);

    float2* g0 = (float2 *)g_;
    for (int i=0;i<ntheta;i++)    
      hipMemcpy(&g0[i*n*pnz], &g[igpu][i*n*pnz], n * pnz * sizeof(float2), hipMemcpyDefault);  
}

void radonusfft::adj(size_t f_, size_t g_, size_t igpu, bool filter) {
    hipSetDevice(igpu);
    float2* g0 = (float2 *)g_;
    for (int i=0;i<ntheta;i++)    
      hipMemcpy(&g[igpu][i*n*pnz],&g0[i*n*pnz], n * pnz * sizeof(float2), hipMemcpyDefault);
    hipMemset(fdee[igpu], 0, (2 * n + 2 * m) * (2 * n + 2 * m) * pnz * sizeof(float2));

    takexy <<<GS2d0, BS2d>>> (x[igpu], y[igpu], theta[igpu], n, ntheta);

    ifftshiftc <<<GS3d3, BS3d>>> (g[igpu], n, ntheta, pnz);
    hipfftExecC2C(plan1d[igpu], (hipfftComplex *)g[igpu], (hipfftComplex *)g[igpu], HIPFFT_FORWARD);
    ifftshiftc <<<GS3d3, BS3d>>> (g[igpu], n, ntheta, pnz);
    if(n%4!=0)
      ifftshiftcmul <<<GS3d3, BS3d>>> (g[igpu], n, ntheta, pnz);
  
    if (filter)
      applyfilter<<<GS3d3, BS3d>>>(g[igpu],n,ntheta,pnz);
    // shift with respect to given center
    shift <<<GS3d3, BS3d>>> (g[igpu], shiftadj[igpu], n, ntheta, pnz);

    gather <<<GS3d3, BS3d>>> (g[igpu], fdee[igpu], x[igpu], y[igpu], m, mu, n, ntheta, pnz, TOMO_ADJ);
    wrap <<<GS3d2, BS3d>>> (fdee[igpu], n, pnz, m, TOMO_ADJ);

    fftshiftc <<<GS3d2, BS3d>>> (fdee[igpu], 2 * n + 2 * m, pnz);
    hipfftExecC2C(plan2d[igpu], (hipfftComplex *)&fdee[igpu][m + m * (2 * n + 2 * m)],
                (hipfftComplex *)&fdee[igpu][m + m * (2 * n + 2 * m)], HIPFFT_BACKWARD);
    fftshiftc <<<GS3d2, BS3d>>> (fdee[igpu], 2 * n + 2 * m, pnz);
    
    divphi <<<GS3d0, BS3d>>> (fdee[igpu], f[igpu], mu, n, pnz, m, TOMO_ADJ);
    //circ <<<GS3d0, BS3d>>> (f, 1.0f / n, n, pnz);
    float2* f0 = (float2 *)f_;
    hipMemcpy(f0, f[igpu], n * n * pnz * sizeof(float2),
              hipMemcpyDefault);
  //}
}
