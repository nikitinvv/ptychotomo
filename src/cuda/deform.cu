#include "deform.cuh"
#include <npp.h>

deform::deform(size_t ntheta, size_t nz, size_t n, size_t ptheta, size_t ngpus) :
 ntheta(ntheta), nz(nz), n(n), ptheta(ptheta), ngpus(ngpus)
{
	cstreams = new hipStream_t[ptheta*ngpus];
	nstreams = new NppStreamContext[ptheta*ngpus];
	for (int igpu=0;igpu<ngpus;igpu++)
	{
		hipSetDevice(igpu);
		for (int i=0;i<ptheta; i++) 
		{
			hipStreamCreate(&cstreams[igpu*ptheta+i]);
			nstreams[igpu*ptheta+i].hStream=cstreams[igpu*ptheta+i];
		}
	}
	hipSetDevice(0);
}

// destructor, memory deallocation
deform::~deform()
{
	free();
}

void deform::free()
{
	if (!is_free)
	{
		for (int igpu=0;igpu<ngpus;igpu++)
		{
			hipSetDevice(igpu);
			for (int i=0;i<ptheta;i++)
			{
				hipStreamDestroy(cstreams[igpu*ptheta+i]);
			}
		}
		delete[] cstreams;		
		delete[] nstreams;		
		
		is_free = true;
		hipSetDevice(0);
	}
}

void deform::remap(size_t g, size_t f, size_t flowx, size_t flowy, size_t gpu)
{
	Npp32f *pSrc = (Npp32f *)f;
	NppiSize oSize = {(int)n,(int)nz};
	Npp32f *pDst = (Npp32f *)g;
	NppiRect oROI = {0,0,(int)n,(int)nz};
	int nStep = 4*n;	
	Npp32f *pXMap = (Npp32f *)flowx;
	Npp32f *pYMap = (Npp32f *)flowy;
	int nXMapStep = 4*n;
	int nYMapStep = 4*n;
	hipSetDevice(gpu);
	for (int i=0;i<ptheta;i++)
	{
		nppiRemap_32f_C1R_Ctx(&pSrc[i*n*nz],oSize,nStep, oROI, &pXMap[i*n*nz], nXMapStep,
			 &pYMap[i*n*nz], nYMapStep, &pDst[i*n*nz], nStep, oSize, NPPI_INTER_LANCZOS,//NPPI_INTER_LANCZOS,//NPPI_INTER_LANCZOS3_ADVANCED,NPPI_INTER_CUBIC
			 nstreams[gpu*ptheta+i]);
		//nppiRemap_32f_C1R (const Npp32f *pSrc, NppiSize oSrcSize, int nSrcStep, NppiRect oSrcROI, const Npp32f *pXMap, int nXMapStep, const Npp32f *pYMap, int nYMapStep, Npp32f *pDst, int nDstStep, NppiSize oDstSizeROI, int eInterpolation)
	}
	hipDeviceSynchronize();
}